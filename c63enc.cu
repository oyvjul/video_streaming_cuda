#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "gpu_data.cuh"
#include "c63_write.h"
#include "c63.h"
#include "tables.cuh"
#include "io.h"
#include "cuda_dct.h"
#include "motion.h"

#include "me.h"

static char *output_file, *input_file;
FILE *outfile;

static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static yuv_t* read_yuv(FILE *file, struct c63_common_cpu *cm)
{
  size_t len = 0;
  yuv_t *image = (yuv_t*) malloc(sizeof(*image));

  /* Read Y. The size of Y is the same as the size of the image. The indices
     represents the color component (0 is Y, 1 is U, and 2 is V) */
  hipHostMalloc((void**)&image->Y, cm->padw[0]*cm->padh[0]);
  len += fread(image->Y, 1, width*height, file);

  /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y
     because (height/2)*(width/2) = (height*width)/4. */
  hipHostMalloc((void**)&image->U, cm->padw[1]*cm->padh[1]);
  len += fread(image->U, 1, (width*height)/4, file);

  /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
  hipHostMalloc((void**)&image->V, cm->padw[2]*cm->padh[2]);
  len += fread(image->V, 1, (width*height)/4, file);

  if (ferror(file))
  {
    perror("ferror");
    exit(EXIT_FAILURE);
  }

  if (feof(file))
  {
	hipHostFree((void*)image->Y);
    hipHostFree((void*)image->U);
	hipHostFree((void*)image->V);
    free(image);

    return NULL;
  }
  else if (len != width*height*1.5)
  {
    fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
    fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);

	hipHostFree((void*)image->Y);
    hipHostFree((void*)image->U);
	hipHostFree((void*)image->V);
    free(image);

    return NULL;
  }

  return image;
}

static void c63_encode_image(struct c63_common_cpu *cpu_cm, struct c63_common_gpu *gpu_cm, uint8_t* d_origY, uint8_t* d_origU, uint8_t* d_origV, uint8_t* d_ref_reconsY, uint8_t* d_ref_reconsU, uint8_t* d_ref_reconsV,
								uint8_t* d_current_reconsY, uint8_t* d_current_reconsU, uint8_t* d_current_reconsV, uint8_t* d_predictedY, uint8_t* d_predictedU, uint8_t* d_predictedV,
									struct macroblock *d_mbsY, struct macroblock *d_mbsU, struct macroblock *d_mbsV, int16_t *d_residualsYDCT, int16_t *d_residualsUDCT, int16_t *d_residualsVDCT)
{
	//zero out macroblock for every frame on the gpu
	hipMemset((void*)d_mbsY, 0, gpu_cm->mb_rows * gpu_cm->mb_cols * sizeof(struct macroblock));
	hipMemset((void*)d_mbsU, 0, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock));
	hipMemset((void*)d_mbsV, 0, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock));

	//zero out macroblock for every frame on the cpu
	memset((void*)cpu_cm->mbs[Y_COMPONENT], 0, gpu_cm->mb_rows * gpu_cm->mb_cols * sizeof(struct macroblock));
	memset((void*)cpu_cm->mbs[U_COMPONENT], 0, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock));
	memset((void*)cpu_cm->mbs[V_COMPONENT], 0, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock));

	for(int i = 0; i < 3; ++i)
		  hipStreamCreate(&streams[i]);


	/* Check if keyframe */
	if (gpu_cm->framenum == 0 || gpu_cm->frames_since_keyframe == gpu_cm->keyframe_interval)
	{
		cpu_cm->keyframe = 1;
		gpu_cm->frames_since_keyframe = 0;

		fprintf(stderr, " (keyframe) ");

		hipMemcpyAsync(d_origY, cpu_cm->orig->Y, gpu_cm->padw[Y_COMPONENT]*gpu_cm->padh[Y_COMPONENT], hipMemcpyHostToDevice, streams[0]);
		hipMemcpyAsync(d_origU, cpu_cm->orig->U, gpu_cm->padw[U_COMPONENT]*gpu_cm->padh[U_COMPONENT], hipMemcpyHostToDevice, streams[1]);
		hipMemcpyAsync(d_origV, cpu_cm->orig->V, gpu_cm->padw[V_COMPONENT]*gpu_cm->padh[V_COMPONENT], hipMemcpyHostToDevice, streams[2]);
	}
	else { cpu_cm->keyframe = 0; }

	if (!cpu_cm->keyframe)
	{
		//set the ref freame to current and send the current frame of the gpu, to the cpu
		d_ref_reconsY = d_current_reconsY;
		d_ref_reconsU = d_current_reconsU;
		d_ref_reconsV = d_current_reconsV;

		hipMemcpyAsync(cpu_cm->ref_recons->Y, d_current_reconsY, gpu_cm->ypw * gpu_cm->yph, hipMemcpyDeviceToHost, streams[0]);
		hipMemcpyAsync(cpu_cm->ref_recons->U, d_current_reconsU, gpu_cm->upw * gpu_cm->uph, hipMemcpyDeviceToHost, streams[1]);
		hipMemcpyAsync(cpu_cm->ref_recons->V, d_current_reconsV, gpu_cm->vpw * gpu_cm->vph, hipMemcpyDeviceToHost, streams[2]);

		hipMemcpyAsync(d_origY, cpu_cm->orig->Y, gpu_cm->padw[Y_COMPONENT]*gpu_cm->padh[Y_COMPONENT], hipMemcpyHostToDevice, streams[0]);
		hipMemcpyAsync(d_origU, cpu_cm->orig->U, gpu_cm->padw[U_COMPONENT]*gpu_cm->padh[U_COMPONENT], hipMemcpyHostToDevice, streams[1]);
		hipMemcpyAsync(d_origV, cpu_cm->orig->V, gpu_cm->padw[V_COMPONENT]*gpu_cm->padh[V_COMPONENT], hipMemcpyHostToDevice, streams[2]);
		
		c63_motion_estimate_new(d_origY, d_origU, d_origV, d_ref_reconsY, d_ref_reconsU,
		d_ref_reconsV, d_mbsY, d_mbsU, d_mbsV, gpu_cm, streams);

		gpu_c63_motion_compensate(d_predictedY, d_predictedU, d_predictedV, d_ref_reconsY, d_ref_reconsU,
		d_ref_reconsV, d_mbsY, d_mbsU, d_mbsV, gpu_cm, streams);
	}

	dct_test(gpu_cm, d_origY, d_origU, d_origV,d_predictedY, d_predictedU, d_predictedV,
			d_residualsYDCT, d_residualsUDCT, d_residualsVDCT, streams,
			cpu_cm->residuals->Ydct, cpu_cm->residuals->Udct, cpu_cm->residuals->Vdct);

	idct_test(gpu_cm, d_predictedY, d_predictedU, d_predictedV, d_residualsYDCT,
			d_residualsUDCT, d_residualsVDCT, d_current_reconsY, d_current_reconsU,
			d_current_reconsV, streams);

	//macroblock can not be sent from device to host concurrently
	hipMemcpyAsync(cpu_cm->mbs[Y_COMPONENT], d_mbsY, gpu_cm->mb_rows*gpu_cm->mb_cols*sizeof(struct macroblock), hipMemcpyDeviceToHost, streams[0]);
	hipMemcpyAsync(cpu_cm->mbs[U_COMPONENT], d_mbsU, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock), hipMemcpyDeviceToHost, streams[0]);
	hipMemcpy(cpu_cm->mbs[V_COMPONENT], d_mbsV, gpu_cm->mb_rows/2 * gpu_cm->mb_cols/2 * sizeof(struct macroblock), hipMemcpyDeviceToHost);

	/* Function dump_image(), found in common.c, can be used here to check if the
		prediction is correct */

	write_frame(cpu_cm);
	++gpu_cm->framenum;
	++gpu_cm->frames_since_keyframe;
}

struct c63_common_gpu* init_c63_enc_gpu(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  struct c63_common_gpu *cm = (c63_common_gpu*) calloc(1, sizeof(struct c63_common_gpu));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->mb_cols = cm->ypw / 8;
  cm->mb_rows = cm->yph / 8;

  /* Quality parameters */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  cm->me_search_range = 16;     // Pixels in every direction
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  return cm;
}

struct c63_common_cpu* init_c63_enc_cpu(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  struct c63_common_cpu *cm = (c63_common_cpu*) calloc(1, sizeof(struct c63_common_cpu));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->mb_cols = cm->ypw / 8;
  cm->mb_rows = cm->yph / 8;

  /* Quality parameters */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  cm->me_search_range = 16;     // Pixels in every direction
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  cm->ref_recons = (yuv_t*) malloc(sizeof(yuv_t));
  cm->ref_recons->Y = (uint8_t*) malloc(cm->ypw * cm->yph);
  cm->ref_recons->U = (uint8_t*) malloc(cm->upw * cm->uph);
  cm->ref_recons->V = (uint8_t*) malloc(cm->vpw * cm->vph);

  cm->residuals = (dct_t*) malloc(sizeof(dct_t));
  hipHostMalloc((void**)&cm->residuals->Ydct, cm->ypw * cm->yph * sizeof(int16_t));
  hipHostMalloc((void**)&cm->residuals->Udct, cm->upw * cm->uph * sizeof(int16_t));
  hipHostMalloc((void**)&cm->residuals->Vdct, cm->vpw * cm->vph * sizeof(int16_t));

  hipHostMalloc((void**)&cm->mbs[Y_COMPONENT], cm->mb_rows * cm->mb_cols * sizeof(struct macroblock));
  hipHostMalloc((void**)&cm->mbs[U_COMPONENT], cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock));
  hipHostMalloc((void**)&cm->mbs[V_COMPONENT], cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock));

  return cm;
}

static void print_help()
{
  printf("%s\n", "Usage: ./c63enc [options] input_file");
  printf("%s\n", "Commandline options:");
  printf("%s\n", "  -h                             Height of images to compress");
  printf("%s\n", "  -w                             Width of images to compress");
  printf("%s\n", "  -o                             Output file (.c63)");
  printf("%s\n", "  [-f]                           Limit number of frames to encode");
  printf("%s\n", "\n");

  exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
  int c;
  uint8_t *d_image_Y = 0;
  uint8_t *d_image_U = 0;
  uint8_t *d_image_V = 0;
  int16_t *d_residuals_YDCT = 0;
  int16_t *d_residuals_UDCT = 0;
  int16_t *d_residuals_VDCT = 0;
  uint8_t *d_curr_rec_Y = 0;
  uint8_t *d_curr_rec_U = 0;
  uint8_t *d_curr_rec_V = 0;
  uint8_t *d_ref_rec_Y = 0;
  uint8_t *d_ref_rec_U = 0;
  uint8_t *d_ref_reconsV = 0;
  uint8_t *d_predicted_Y = 0;
  uint8_t *d_predicted_U = 0;
  uint8_t *d_predicted_V = 0;
  struct macroblock *d_mbY = 0;
  struct macroblock *d_mbU = 0;
  struct macroblock *d_mbV = 0;

  if (argc == 1) { print_help(); }

  while ((c = getopt(argc, argv, "h:w:o:f:i:")) != -1)
  {
    switch (c)
    {
      case 'h':
        height = atoi(optarg);
        break;
      case 'w':
        width = atoi(optarg);
        break;
      case 'o':
        output_file = optarg;
        break;
      case 'f':
        limit_numframes = atoi(optarg);
        break;
      default:
        print_help();
        break;
    }
  }

  if (optind >= argc)
  {
    fprintf(stderr, "Error getting program options, try --help.\n");
    exit(EXIT_FAILURE);
  }

  outfile = fopen(output_file, "wb");

  if (outfile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  struct c63_common_cpu *cpu_cm = init_c63_enc_cpu(width, height);
  cpu_cm->e_ctx.fp = outfile;
  struct c63_common_gpu *gpu_cm = init_c63_enc_gpu(width, height);
  gpu_cm->e_ctx.fp = outfile;

  input_file = argv[optind];

  if (limit_numframes) { printf("Limited to %d frames.\n", limit_numframes); }

  FILE *infile = fopen(input_file, "rb");

  if (infile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  int numframes = 0;

  //Allocate GPU Data
  hipMalloc((void**)&d_ref_rec_Y, cpu_cm->ypw * cpu_cm->yph);
  hipMalloc((void**)&d_ref_rec_U, cpu_cm->upw * cpu_cm->uph);
  hipMalloc((void**)&d_ref_reconsV, cpu_cm->vpw * cpu_cm->vph);

  hipMalloc((void**)&d_curr_rec_Y, cpu_cm->ypw * cpu_cm->yph);
  hipMalloc((void**)&d_curr_rec_U, cpu_cm->upw * cpu_cm->uph);
  hipMalloc((void**)&d_curr_rec_V, cpu_cm->vpw * cpu_cm->vph);

  hipMalloc((void**)&d_mbY, cpu_cm->mb_rows * cpu_cm->mb_cols * sizeof(struct macroblock));
  hipMalloc((void**)&d_mbU, cpu_cm->mb_rows/2 * cpu_cm->mb_cols/2 * sizeof(struct macroblock));
  hipMalloc((void**)&d_mbV, cpu_cm->mb_rows/2 * cpu_cm->mb_cols/2 * sizeof(struct macroblock));

  hipMalloc((void**)&d_image_Y, cpu_cm->padw[Y_COMPONENT]*cpu_cm->padh[Y_COMPONENT]);
  hipMalloc((void**)&d_image_U, cpu_cm->padw[U_COMPONENT]*cpu_cm->padh[U_COMPONENT]);
  hipMalloc((void**)&d_image_V, cpu_cm->padw[V_COMPONENT]*cpu_cm->padh[V_COMPONENT]);

  hipMalloc((void**)&d_predicted_Y, cpu_cm->ypw * cpu_cm->yph);
  hipMalloc((void**)&d_predicted_U, cpu_cm->upw * cpu_cm->uph);
  hipMalloc((void**)&d_predicted_V, cpu_cm->vpw * cpu_cm->vph);

  hipMalloc((void**)&d_residuals_YDCT, cpu_cm->ypw * cpu_cm->yph * sizeof(int16_t));
  hipMalloc((void**)&d_residuals_UDCT, cpu_cm->upw * cpu_cm->uph * sizeof(int16_t));
  hipMalloc((void**)&d_residuals_VDCT, cpu_cm->vpw * cpu_cm->vph * sizeof(int16_t));

  hipMemset((void*)d_predicted_Y, 0, cpu_cm->ypw * cpu_cm->yph * sizeof(uint8_t));
  hipMemset((void*)d_predicted_U, 0, cpu_cm->upw * cpu_cm->uph * sizeof(uint8_t));
  hipMemset((void*)d_predicted_V, 0, cpu_cm->vpw * cpu_cm->vph * sizeof(uint8_t));

  hipMemset((void*)d_residuals_YDCT, 0, cpu_cm->ypw * cpu_cm->yph * sizeof(int16_t));
  hipMemset((void*)d_residuals_UDCT, 0, cpu_cm->upw * cpu_cm->uph * sizeof(int16_t));
  hipMemset((void*)d_residuals_VDCT, 0, cpu_cm->vpw * cpu_cm->vph * sizeof(int16_t));

  while (1)
  {
    cpu_cm->orig = read_yuv(infile, cpu_cm);

	if (!cpu_cm->orig) { break; }

    printf("Encoding frame %d, ", numframes);
	c63_encode_image(cpu_cm, gpu_cm, d_image_Y, d_image_U, d_image_V, d_ref_rec_Y,
			d_ref_rec_U, d_ref_reconsV, d_curr_rec_Y, d_curr_rec_U, d_curr_rec_V,
			d_predicted_Y, d_predicted_U, d_predicted_V, d_mbY, d_mbU, d_mbV,
			d_residuals_YDCT, d_residuals_UDCT, d_residuals_VDCT);

	hipHostFree((void*)cpu_cm->orig->Y);
	hipHostFree((void*)cpu_cm->orig->U);
	hipHostFree((void*)cpu_cm->orig->V);
    free(cpu_cm->orig);

    printf("%s\n", "Done!");

    ++numframes;

    if (limit_numframes && numframes >= limit_numframes) { break; }
  }

  hipFree((void*)d_ref_rec_Y);
  hipFree((void*)d_ref_rec_U);
  hipFree((void*)d_ref_reconsV);

  hipFree((void*)d_curr_rec_Y);
  hipFree((void*)d_curr_rec_U);
  hipFree((void*)d_curr_rec_V);

  hipFree((void*)d_mbY);
  hipFree((void*)d_mbU);
  hipFree((void*)d_mbV);

  hipFree((void*)d_image_Y);
  hipFree((void*)d_image_U);
  hipFree((void*)d_image_V);

  hipFree((void*)d_predicted_Y);
  hipFree((void*)d_predicted_U);
  hipFree((void*)d_predicted_V);

  hipFree((void*)d_residuals_YDCT);
  hipFree((void*)d_residuals_UDCT);
  hipFree((void*)d_residuals_VDCT);

  free(cpu_cm->ref_recons->Y);
  free(cpu_cm->ref_recons->U);
  free(cpu_cm->ref_recons->V);
  free(cpu_cm->ref_recons);

  hipHostFree((void*)cpu_cm->residuals->Ydct);
  hipHostFree((void*)cpu_cm->residuals->Udct);
  hipHostFree((void*)cpu_cm->residuals->Vdct);
  free(cpu_cm->residuals);

  hipHostFree((void*)cpu_cm->mbs[Y_COMPONENT]);
  hipHostFree((void*)cpu_cm->mbs[U_COMPONENT]);
  hipHostFree((void*)cpu_cm->mbs[V_COMPONENT]);

  int i;
  for(i = 0; i < 3; ++i)
  {
	  hipStreamDestroy(streams[i]);
  }

  fclose(outfile);
  fclose(infile);

  hipDeviceReset();

  return EXIT_SUCCESS;
}

